#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <system_error>
#include "../../include/kernels.h"
#include "hip/hip_fp16.h"
#include "hip/hip_runtime.h"
#include "helper.h"
#include "math.h"
#include "stdio.h"

struct GpuTimer
{
    hipStream_t _stream_id;
    hipEvent_t _start;
    hipEvent_t _stop;

    /// Constructor
    GpuTimer() : _stream_id(0)
    {
        CUDA_CHECK(hipEventCreate(&_start));
        CUDA_CHECK(hipEventCreate(&_stop));
    }

    /// Destructor
    ~GpuTimer()
    {
        CUDA_CHECK(hipEventDestroy(_start));
        CUDA_CHECK(hipEventDestroy(_stop));
    }

    /// Start the timer for a given stream (defaults to the default stream)
    void start(hipStream_t stream_id = 0)
    {
        _stream_id = stream_id;
        CUDA_CHECK(hipEventRecord(_start, _stream_id));
    }

    /// Stop the timer
    void stop()
    {
        CUDA_CHECK(hipEventRecord(_stop, _stream_id));
    }

    /// Return the elapsed time (in milliseconds)
    float elapsed_millis()
    {
        float elapsed = 0.0;
        CUDA_CHECK(hipEventSynchronize(_stop));
        CUDA_CHECK(hipEventElapsedTime(&elapsed, _start, _stop));
        return elapsed;
    }
};

int GetSizeofDtype(DataType dtype) {
    if (dtype == DataType::INT32 || dtype == DataType::FLOAT32) {
        return 4;
    } else if (dtype == DataType::FLOAT16 || dtype == DataType::HALF) {
        return 2;
    } else if (dtype == DataType::INT64) {
        return 8;
    } else if (dtype == DataType::INT8 || dtype == DataType::BOOL) {
        return 1;
    }
    return 1;
}

size_t GetProdofVector(std::vector<int> shapes) {
    size_t sum = 1;
    for (auto& shape : shapes) {
        sum *= shape;
    }
    return sum;
}

__global__ void convert_fp32_to_fp16_cuda(float* in_ptr, half* out_ptr, int length) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length) {
        out_ptr[index] = __float2half(in_ptr[index]);
    }
}

__global__ void convert_fp16_to_fp32_cuda(half* in_ptr, float* out_ptr, int length) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length) {
        out_ptr[index] = __half2float(in_ptr[index]);
    }
}

YTensor::YTensor() {}

YTensor::~YTensor() {
    Free();
}

bool YTensor::Malloc(Dims dims, DataType dtype, DataLayout layout) {
    this->sizeoftype = GetSizeofDtype(dtype);
    this->length = GetProdofVector(dims.shapes);
    this->cpu_ptr = malloc(this->sizeoftype * this->length);
    this->data = cpu_ptr;
    this->gpu_ptr = nullptr;
    this->nb_dims = dims.nb_dims;
    this->shape = dims.shapes;
    this->dtype = dtype;
    this->layout = layout;
    this->is_gpu = false;
    return true;
}

bool YTensor::Free() {
    if (this->cpu_ptr) {
        free(this->cpu_ptr);
        this->cpu_ptr = nullptr;
    }
    if (this->gpu_ptr) {
        hipFree(this->gpu_ptr);
        this->gpu_ptr = nullptr;
    }
    this->data = nullptr;
    return true;
}

bool YTensor::Zeros(Dims dims, DataType dtype, DataLayout layout) {
    this->sizeoftype = GetSizeofDtype(dtype);
    this->length = GetProdofVector(dims.shapes);
    this->cpu_ptr = malloc(this->sizeoftype * this->length);
    if (not this->cpu_ptr) {
        std::cout << "malloc error \n";
    }
    memset(this->cpu_ptr, 0, this->sizeoftype * this->length);
    this->data = cpu_ptr;
    this->gpu_ptr = nullptr;
    this->nb_dims = dims.nb_dims;
    this->shape = dims.shapes;
    this->dtype = dtype;
    this->layout = layout;
    this->is_gpu = false;
    return true;
}

bool YTensor::Float() {
    if (!this->is_gpu) {
        printf("[Error] data not on gpu !!\n");
        return false;
    }
    if (this->dtype == DataType::HALF) {
        this->dtype = DataType::HALF;
        this->sizeoftype = sizeof(float);
        int block_size = 512;
        int grid_size = (this->length + block_size - 1) / block_size;
        void* tmp;
        hipMalloc((void**)&tmp, this->length * sizeof(half));
        hipMemcpy(tmp, this->gpu_ptr, this->length * sizeof(half), hipMemcpyDeviceToDevice);
        hipMalloc((void**)&(this->gpu_ptr), this->length * this->sizeoftype);
        convert_fp16_to_fp32_cuda<<<grid_size, block_size>>>((half*)tmp, (float*)this->gpu_ptr,
                                                             this->length);
        this->data = this->gpu_ptr;
        hipFree(tmp);
    } else if (this->dtype == DataType::FLOAT32) {
        return true;
    } else {
        printf("[Error] datatype not correct !!\n");
        return false;
    }
    return true;
}

bool YTensor::Half() {
    if (!this->is_gpu) {
        printf("[Error] data not on gpu !!\n");
        return false;
    }
    if (this->dtype == DataType::FLOAT32) {
        this->dtype = DataType::FLOAT32;
        this->sizeoftype = sizeof(half);
        int block_size = 512;
        int grid_size = (this->length + block_size - 1) / block_size;
        void* tmp;
        hipMalloc((void**)&tmp, this->length * sizeof(float));
        hipMemcpy(tmp, this->gpu_ptr, this->length * sizeof(float), hipMemcpyDeviceToDevice);
        hipMalloc((void**)&(this->gpu_ptr), this->length * this->sizeoftype);
        convert_fp32_to_fp16_cuda<<<grid_size, block_size>>>((float*)tmp, (half*)this->gpu_ptr,
                                                             this->length);
        this->data = this->gpu_ptr;
        hipFree(tmp);
    } else if (this->dtype == DataType::HALF) {
        return true;
    } else {
        printf("[Error] datatype not correct !!\n");
        return false;
    }
    return true;
}

bool YTensor::CUDA() {
    if (this->is_gpu) {
        checkCudaStatus(hipMemcpy(this->gpu_ptr, this->cpu_ptr, this->sizeoftype * this->length,
                                   hipMemcpyHostToDevice));
    } else {
        checkCudaStatus(hipMalloc((void**)&(this->gpu_ptr), this->sizeoftype * this->length));
        checkCudaStatus(hipMemcpy(this->gpu_ptr, this->cpu_ptr, this->sizeoftype * this->length,
                                   hipMemcpyHostToDevice));
    }
    this->is_gpu = true;
    this->data = this->gpu_ptr;
    return true;
}

bool YTensor::CPU() {
    if (this->is_gpu) {
        checkCudaStatus(hipMemcpy(this->cpu_ptr, this->gpu_ptr, this->sizeoftype * this->length,
                                   hipMemcpyDeviceToHost));
        checkCudaStatus(hipFree(this->gpu_ptr));
    }
    this->is_gpu = false;
    this->data = this->cpu_ptr;
    this->gpu_ptr = nullptr;
    return true;
}

void YTensor::SetDataPtr(int64_t ptr) {
    if (this->is_gpu) {
        this->data = (void*)ptr;
        this->gpu_ptr = (void*)ptr;
    } else {
        this->data = (void*)ptr;
        this->cpu_ptr = (void*)ptr;
    }
}

int64_t YTensor::GetDataPtr() {
    return (int64_t)(this->data);
}

void YTensor::SetShape(std::vector<int> dims) {
    this->shape = dims;
}

std::vector<int> YTensor::GetShape() {
    return this->shape;
}


void* create_handle() {
    Handles* handle = new Handles();
    return (void*)handle;
}
