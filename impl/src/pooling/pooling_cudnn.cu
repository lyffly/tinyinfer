#include <cudnn_v9.h>
#include <iostream>
#include "../../include/kernels.h"
#include "hipblaslt.h"
#include "hipblas.h"
#include "hip/hip_fp16.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include "stdio.h"

#include <cstdint>
#include <cstdio>
#include <string>
#include <vector>
#include "../helper/helper.h"

static hipdnnHandle_t cudnn_handle;

void* create_pooling_desc() {
    PoolDesc* desc = new PoolDesc();
    hipdnnCreateTensorDescriptor(&desc->input_desc);
    hipdnnCreateTensorDescriptor(&desc->output_desc);
    hipdnnCreatePoolingDescriptor(&desc->pooling_desc);
    return (void*)desc;
}

void setup_pooling_descriptor(std::vector<int>& kernels, std::vector<int>& paddings,
                              std::vector<int>& strides, std::vector<int>& in_shape,
                              std::vector<int>& out_shape, std::string optype, std::string dtype,
                              std::string layout, void* desc) {
    PoolDesc* desc_ = (PoolDesc*)desc;
    hipdnnDataType_t infer_data_type;
    hipdnnTensorFormat_t infer_layout;
    if (layout == "nchw") {
        infer_layout = HIPDNN_TENSOR_NCHW;
    } else if (layout == "nhwc") {
        infer_layout = HIPDNN_TENSOR_NHWC;
    }
    if (dtype == "float16") {
        infer_data_type = HIPDNN_DATA_HALF;
    } else if (dtype == "float32") {
        infer_data_type = HIPDNN_DATA_FLOAT;
    }

    int batch = in_shape.at(0);
    int inc = in_shape.at(1);
    int inh = in_shape.at(2);
    int inw = in_shape.at(3);

    int outc = out_shape.at(1);
    int outh = out_shape.at(2);
    int outw = out_shape.at(3);
    // printf("input shape : %d %d %d %d \n", batch, inc, inh, inw);
    // printf("output shape : %d %d %d %d \n", batch, outc, outh, outw);

    hipdnnSetTensor4dDescriptor(desc_->input_desc, infer_layout, infer_data_type, batch, inc, inh,
                               inw);
    hipdnnSetTensor4dDescriptor(desc_->output_desc, infer_layout, infer_data_type, batch, outc, outh,
                               outw);

    hipdnnPoolingMode_t mode;
    if (optype == "GlobalAveragePool") {
        mode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
        hipdnnNanPropagation_t maxpoolingNanOpt = HIPDNN_PROPAGATE_NAN;
        hipdnnSetPooling2dDescriptor(desc_->pooling_desc, mode, maxpoolingNanOpt, in_shape.at(2),
                                    in_shape.at(3), 0, 0, 1, 1);
    } else if (optype == "MaxPool") {
        mode = HIPDNN_POOLING_MAX;
        hipdnnNanPropagation_t maxpoolingNanOpt = HIPDNN_PROPAGATE_NAN;
        hipdnnSetPooling2dDescriptor(desc_->pooling_desc, mode, maxpoolingNanOpt, kernels.at(0),
                                    kernels.at(1), paddings.at(0), paddings.at(1), strides.at(0),
                                    strides.at(1));
    }
}


bool pooling_cudnn_backend(int64_t in_ptr, int64_t out_ptr, std::vector<int> kernels,
                     std::vector<int> paddings, std::vector<int> strides, std::vector<int> in_shape,
                     std::vector<int> out_shape, std::string optype, std::string dtype,
                     std::string layout, int64_t pstream, void* desc) {

    // setup_pooling_descriptor(kernels, paddings, strides, in_shape, out_shape, optype, dtype, layout, desc);

    PoolDesc* desc_ = (PoolDesc*)desc;
    if (!cudnn_handle) {
        hipdnnCreate(&cudnn_handle);
        hipdnnSetStream(cudnn_handle, (hipStream_t)pstream);
    }

    float alpha_ = 1.0f;
    float beta_ = 0.0f;

    hipdnnStatus_t Error =
        hipdnnPoolingForward(cudnn_handle, desc_->pooling_desc, &alpha_, desc_->input_desc,
                            (void*)in_ptr, &beta_, desc_->output_desc, (void*)out_ptr);

    if (Error != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr, "[Error] cudnn forward failed!\n");
    }
    return true;
}
