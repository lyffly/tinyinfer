#include "hip/hip_runtime.h"
#include "../../include/kernels.h"
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
#include "stdio.h"
#include "math.h"
#include <iostream>

#include <cstdint>
#include <vector>
#include <string>
#include <cstdio>

enum CastOpType {
    Cast_Unknown = 0,
    Cast_FP32_FP16,
    Cast_FP16_FP32,
    Cast_FP32_INT8,
    Cast_INT8_FP32,
};

template<typename T1, typename T2>
__device__ inline T2 cast_op(T1 input);


template<> __device__ inline float cast_op<half, float>(half input) {
    return __float2half(input);
}
template<> __device__ inline half cast_op<float, half>(float input) {
    return __half2float(input);
}

template<typename T1, typename T2>
__global__ void cast_fp_cuda(T1 *in, T2 *out, int length) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    auto in_ptr = reinterpret_cast<T1*>(in);
    auto out_ptr = reinterpret_cast<T2*>(out);
    if (index < length) {
        out_ptr[index] = cast_op<T1, T2>(in_ptr[index]);
    }
}


bool cast_backend(int64_t in_ptr, int64_t out_ptr, std::vector<int> in_shape, std::vector<int> out_shape, 
                    std::string layout, std::string in_dtype, std::string out_dtype, int64_t pstream) {
    hipStream_t stream = (hipStream_t)pstream;
    int block_size = 512;
    int length = 1;
    for (auto& shape : out_shape) {
        length *= shape;
    }
    // toto deal with broad cast and layout
    int grid_size = (length + block_size - 1) / block_size;

    if (in_dtype == "float16" and out_dtype == "float32") {
        cast_fp_cuda<half, float><<<grid_size, block_size,0, stream>>>((half*)in_ptr, (float*)out_ptr,
                                            (int)length);
    } else if (in_dtype == "float32" and out_dtype == "float16"){
        cast_fp_cuda<float, half><<<grid_size, block_size,0, stream>>>((float*)in_ptr, (__half*)out_ptr,
                                            (int)length);
    } 
    
    return true;
}
