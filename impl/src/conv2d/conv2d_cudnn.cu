#include "hip/hip_runtime.h"
#include <cudnn_v9.h>
#include <iostream>
#include "../../include/kernels.h"
#include "hipblaslt.h"
#include "hipblas.h"
#include "hip/hip_fp16.h"
#include "hip/hip_runtime.h"
#include "hipDNN.h"
#include "math.h"
#include "stdio.h"

#include <cstdint>
#include <cstdio>
#include <string>
#include <vector>
#include "../helper/helper.h"

static hipdnnHandle_t cudnn_handle;

void* create_conv2d_desc() {
    ConvDesc* desc = new ConvDesc();
    hipdnnCreateTensorDescriptor(&desc->input_desc);
    hipdnnCreateTensorDescriptor(&desc->output_desc);
    hipdnnCreateFilterDescriptor(&desc->kernel_desc);
    hipdnnCreateTensorDescriptor(&desc->bias_desc);
    hipdnnCreateConvolutionDescriptor(&desc->conv_desc);
    hipdnnCreateActivationDescriptor(&desc->activation_desc);
    hipdnnCreateActivationDescriptor(&desc->activation_desc);
    return (void*)desc;
}

void setup_descriptor(std::vector<int>& kernels, std::vector<int>& paddings,
                      std::vector<int>& strides, std::vector<int>& dilations, int group,
                      std::vector<int>& in_shape, std::vector<int>& weight_shape,
                      std::vector<int>& bias_shape, std::vector<int>& out_shape, std::string dtype,
                      std::string layout, void* desc) {
    ConvDesc* desc_ = (ConvDesc*)desc;
    hipdnnDataType_t infer_data_type;
    hipdnnTensorFormat_t infer_layout;
    if (layout == "nchw") {
        infer_layout = HIPDNN_TENSOR_NCHW;
    } else if (layout == "nhwc") {
        infer_layout = HIPDNN_TENSOR_NHWC;
    }
    if (dtype == "float16") {
        infer_data_type = HIPDNN_DATA_HALF;
    } else if (dtype == "float32") {
        infer_data_type = HIPDNN_DATA_FLOAT;
    }

    int batch = in_shape.at(0);
    int inc = in_shape.at(1);
    int inh = in_shape.at(2);
    int inw = in_shape.at(3);

    int outc = out_shape.at(1);
    int outh = out_shape.at(2);
    int outw = out_shape.at(3);
    // printf("input shape : %d %d %d %d \n", batch, inc, inh, inw);
    // printf("output shape : %d %d %d %d \n", batch, outc, outh, outw);

    hipdnnSetTensor4dDescriptor(desc_->input_desc, infer_layout, infer_data_type, batch, inc, inh,
                               inw);
    hipdnnSetTensor4dDescriptor(desc_->output_desc, infer_layout, infer_data_type, batch, outc, outh,
                               outw);
    hipdnnSetFilter4dDescriptor(desc_->kernel_desc, infer_data_type, infer_layout, outc, inc,
                               kernels.at(0), kernels.at(1));

    // printf("kernel shape : %d %d %d %d \n", outc, inc, kernels.at(0), kernels.at(1));

    hipdnnSetTensor4dDescriptor(desc_->bias_desc,
                               /*format=*/infer_layout,
                               /*dataType=*/infer_data_type,
                               /*batch_size=*/1,
                               /*channels=*/outc,
                               /*image_height=*/1,
                               /*image_width=*/1);
    // printf("bias shape : %d %d %d %d \n", 1, outc, 1, 1);

    hipdnnSetConvolutionMathType(desc_->conv_desc, HIPDNN_TENSOR_OP_MATH);

    hipdnnSetConvolution2dDescriptor(desc_->conv_desc, paddings.at(0), paddings.at(1), strides.at(0),
                                    strides.at(1), dilations.at(0), dilations.at(1),
                                    HIPDNN_CROSS_CORRELATION, infer_data_type);

    // printf("padding : %d %d, stride: %d %d, dilation: %d %d \n", paddings.at(0), paddings.at(1),
    //        strides.at(0), strides.at(1), dilations.at(0), dilations.at(1));
    hipdnnSetConvolutionGroupCount(desc_->conv_desc, group);
    hipdnnSetActivationDescriptor(desc_->activation_desc, HIPDNN_ACTIVATION_PATHTRU,
                                 HIPDNN_PROPAGATE_NAN, 0);
}

int64_t get_conv2d_algo(std::vector<int> kernels, std::vector<int> paddings,
                        std::vector<int> strides, std::vector<int> dilations, int group,
                        std::vector<int> in_shape, std::vector<int> weight_shape,
                        std::vector<int> bias_shape, std::vector<int> out_shape, std::string dtype,
                        std::string layout, int64_t pstream, void* desc) {
    ConvDesc* desc_ = (ConvDesc*)desc;
    if (!cudnn_handle) {
        hipdnnCreate(&cudnn_handle);
        hipdnnSetStream(cudnn_handle, (hipStream_t)pstream);
    }

    setup_descriptor(kernels, paddings, strides, dilations, group, in_shape, weight_shape,
                     bias_shape, out_shape, dtype, layout, desc_);

    hipdnnConvolutionFwdAlgoPerf_t perfResults[10];
    int returnedAlgoCount = 0;
    // cudnnGetConvolutionForwardAlgorithm_v7(cudnn_handle, input_desc, kernel_desc, conv_desc,
    //                                        output_desc, 1, &returnedAlgoCount, perfResults);
    hipdnnFindConvolutionForwardAlgorithm(cudnn_handle, desc_->input_desc, desc_->kernel_desc,
                                         desc_->conv_desc, desc_->output_desc, 2,
                                         &returnedAlgoCount, perfResults);

    // printf("find algo: %d, math %d \n\n ", int32_t(perfResults[0].algo),
    //        int32_t(perfResults[0].mathType));

    // return int64_t(HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);
    if (returnedAlgoCount > 0) {
        return int64_t(perfResults[0].algo);
    } else {
        return int64_t(-1);
    }
}

int64_t get_conv2d_workspace_size(std::vector<int> kernels, std::vector<int> paddings,
                                  std::vector<int> strides, std::vector<int> dilations, int group,
                                  std::vector<int> in_shape, std::vector<int> weight_shape,
                                  std::vector<int> bias_shape, std::vector<int> out_shape,
                                  std::string dtype, std::string layout, int64_t algo,
                                  int64_t pstream, void* desc) {
    ConvDesc* desc_ = (ConvDesc*)desc;
    if (!cudnn_handle) {
        hipdnnCreate(&cudnn_handle);
        hipdnnSetStream(cudnn_handle, (hipStream_t)pstream);
    }

    size_t space_size = 0;
    hipdnnConvolutionFwdAlgo_t algo_ = (hipdnnConvolutionFwdAlgo_t)algo;

    hipdnnStatus_t Error = hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn_handle, desc_->input_desc, desc_->kernel_desc, desc_->conv_desc, desc_->output_desc,
        algo_, &space_size);
    if (Error != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr, "[Error] cudnn get workspace size failed!\n");
    }

    return int64_t(space_size);
}

// batch*c*hw + 1*c*1 = batch*c*hw
template <typename T>
__global__ void add_conv2d_bias_fp(T* inout, T* bias, int batch, int c, int hw, int length) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    auto inout_ptr = reinterpret_cast<T*>(inout);
    auto bias_ptr = reinterpret_cast<T*>(bias);
    if ((index < length)) {
        int index_bias = index % (c * hw) / hw;
        inout_ptr[index] = inout_ptr[index] + bias_ptr[index_bias];
    }
}

bool conv2d_backend(int64_t in_ptr, int64_t weight_ptr, int64_t bias_ptr, int64_t out_ptr,
                    int64_t workspace_size, int64_t workspace_ptr, int64_t algo,
                    std::vector<int> kernels, std::vector<int> paddings, std::vector<int> strides,
                    std::vector<int> dilations, int group, std::vector<int> in_shape,
                    std::vector<int> weight_shape, std::vector<int> bias_shape,
                    std::vector<int> out_shape, std::string dtype, std::string layout,
                    int64_t pstream, void* desc) {
    ConvDesc* desc_ = (ConvDesc*)desc;
    if (!cudnn_handle) {
        hipdnnCreate(&cudnn_handle);
        hipdnnSetStream(cudnn_handle, (hipStream_t)pstream);
    }

    size_t space_size_ = (size_t)workspace_size;
    hipdnnConvolutionFwdAlgo_t algo_ = (hipdnnConvolutionFwdAlgo_t)algo;

    float alpha_ = 1.0f;
    float beta_ = 0.0f;
    // conv
    // {
    //     // hipdnnStatus_t Error = hipdnnConvolutionForward(cudnn_handle,
    //     hipdnnConvolutionForward(cudnn_handle, &alpha_, input_desc, (void*)in_ptr, kernel_desc,
    //                             (void*)weight_ptr, conv_desc, algo_, (void*)workspace_ptr,
    //                             space_size_, &beta_, output_desc, (void*)out_ptr);
    //     // if (Error != HIPDNN_STATUS_SUCCESS) {
    //     //   fprintf(stderr, "[Error] cudnn forward failed!\n");
    //     // }

    //     // add bias
    //     int block_size = 512;
    //     int batch = out_shape.at(0);
    //     int outc = out_shape.at(1);
    //     int hw = out_shape.at(2) * out_shape.at(3);
    //     int length = batch * outc * hw;
    //     hipStream_t stream = (hipStream_t)pstream;
    //     int grid_size = (length + block_size - 1) / block_size;
    //     if (dtype == "float32") {
    //         add_conv2d_bias_fp<float><<<grid_size, block_size, 0, stream>>>(
    //             (float*)out_ptr, (float*)bias_ptr, batch, outc, hw, length);
    //     } else if (dtype == "float16") {
    //         add_conv2d_bias_fp<half><<<grid_size, block_size, 0, stream>>>(
    //             (half*)out_ptr, (half*)bias_ptr, batch, outc, hw, length);
    //     }
    // }

    // conv bias activation
    {
        // printf("[conv] algo %d, space size %ld, ptr %p \n\n", (int)algo_, space_size_,
        //        (void*)workspace_ptr);

        hipdnnStatus_t Error = cudnnConvolutionBiasActivationForward(
            cudnn_handle, &alpha_, desc_->input_desc, (void*)in_ptr, desc_->kernel_desc,
            (void*)weight_ptr, desc_->conv_desc, algo_, (void*)workspace_ptr, space_size_, &beta_,
            desc_->output_desc, (void*)out_ptr, desc_->bias_desc, (void*)bias_ptr,
            desc_->activation_desc, desc_->output_desc, (void*)out_ptr);

        if (Error != HIPDNN_STATUS_SUCCESS) {
            fprintf(stderr, "[Error] cudnn forward failed!\n");
        }
    }
    return true;
}
