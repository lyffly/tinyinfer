#include "../../include/kernels.h"
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
#include "hipblaslt.h"
#include "hipblas.h"
#include "hipDNN.h"
#include "stdio.h"
#include "math.h"
#include <cudnn_ops_infer.h>
#include <iostream>

#include <cstdint>
#include <vector>
#include <string>
#include <cstdio>
#include "../helper/helper.h"

static hipdnnHandle_t cudnn_handle;

void setup_descriptor_nchw(std::vector<int> &kernels, std::vector<int> &paddings, std::vector<int> &strides, std::vector<int> &dilations, int group,
                std::vector<int> &in_shape, std::vector<int> &weight_shape, std::vector<int> &bias_shape, std::vector<int> &out_shape, 
                std::string dtype, std::string layout, hipdnnTensorDescriptor_t &input_desc,
                hipdnnTensorDescriptor_t &output_desc, hipdnnFilterDescriptor_t &kernel_desc, hipdnnConvolutionDescriptor_t &conv_desc) {
    int batch = in_shape.at(0);
    int inc = in_shape.at(1);
    int inh = in_shape.at(2);
    int inw = in_shape.at(3);

    int outc = out_shape.at(1);
    int outh = out_shape.at(2);
    int outw = out_shape.at(3);

    if (!cudnn_handle) hipdnnCreate(&cudnn_handle);

	hipdnnCreateTensorDescriptor(&input_desc);
    if (dtype=="float16") {
        hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_HALF,
            batch, inc, inh, inw);
    } else if (dtype=="float32") {
        hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
            batch, inc, inh, inw);
    }

	hipdnnCreateTensorDescriptor(&output_desc);
    if (dtype=="float16") {
        hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_HALF,
            batch, outc, outh, outw);
    } else if (dtype=="float32") {
        hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
            batch, outc, outh, outw);
    }

	hipdnnCreateFilterDescriptor(&kernel_desc);
    if (dtype=="float16") {
        hipdnnSetFilter4dDescriptor(kernel_desc, HIPDNN_DATA_HALF, HIPDNN_TENSOR_NCHW,
	        outc, inc, kernels.at(0), kernels.at(1));
    } else if (dtype=="float32") {
        hipdnnSetFilter4dDescriptor(kernel_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
	        outc, inc, kernels.at(0), kernels.at(1));
    }

	hipdnnCreateConvolutionDescriptor(&conv_desc);
    if (dtype=="float16") {
        hipdnnSetConvolution2dDescriptor(conv_desc, paddings.at(0), paddings.at(1),
            strides.at(0), strides.at(1),
            dilations.at(0), dilations.at(1), 
            HIPDNN_CONVOLUTION, HIPDNN_DATA_HALF);
    } else if (dtype=="float32") {
        hipdnnSetConvolution2dDescriptor(conv_desc, paddings.at(0), paddings.at(1),
            strides.at(0), strides.at(1),
            dilations.at(0), dilations.at(1), 
            HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);
    }
    hipdnnSetConvolutionGroupCount(conv_desc, group);
    hipdnnMathType_t math_type = CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION;
    cudnnGetConvolutionMathType(conv_desc, &math_type);

}

int64_t get_conv2d_algo(std::vector<int> kernels, std::vector<int> paddings, std::vector<int> strides, 
                std::vector<int> dilations, int group,
                std::vector<int> in_shape, std::vector<int> weight_shape, std::vector<int> bias_shape, std::vector<int> out_shape, 
                std::string dtype, std::string layout) {
    if (!cudnn_handle) hipdnnCreate(&cudnn_handle);

    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t output_desc;
    hipdnnFilterDescriptor_t kernel_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
	setup_descriptor_nchw(kernels, paddings, strides, dilations, group, in_shape, weight_shape, bias_shape, out_shape, 
                dtype, layout,input_desc,output_desc, kernel_desc, conv_desc);

    hipdnnConvolutionFwdAlgoPerf_t perfResults[10];
    int returnedAlgoCount = 0;
    cudnnGetConvolutionForwardAlgorithm_v7(
        cudnn_handle,
        input_desc,
        kernel_desc,
        conv_desc,
        output_desc,
        10,
        &returnedAlgoCount,
        perfResults);
    if (returnedAlgoCount>0) {
        return int64_t(perfResults[0].algo);
    } else {
        return int64_t(-1);
    }
}

int64_t get_conv2d_workspace_size(std::vector<int> kernels, std::vector<int> paddings, std::vector<int> strides, 
                std::vector<int> dilations, int group,
                std::vector<int> in_shape, std::vector<int> weight_shape, std::vector<int> bias_shape, std::vector<int> out_shape, 
                std::string dtype, std::string layout, int64_t algo) {
    if (!cudnn_handle) hipdnnCreate(&cudnn_handle);

    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t output_desc;
    hipdnnFilterDescriptor_t kernel_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
	setup_descriptor_nchw(kernels, paddings, strides, dilations, group, in_shape, weight_shape, bias_shape, out_shape, 
                dtype, layout,input_desc,output_desc, kernel_desc, conv_desc);
	
    size_t space_size = 0;
    hipdnnConvolutionFwdAlgo_t algo_ = (hipdnnConvolutionFwdAlgo_t)algo;
    
    hipdnnStatus_t Error = hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle, input_desc,kernel_desc, conv_desc, output_desc, 
        algo_, &space_size);
    if (Error != HIPDNN_STATUS_SUCCESS) {
		fprintf(stderr, "[Error] cudnn get workspace size failed!\n");
	}

    return int64_t(space_size);
}


bool conv2d_backend(int64_t in_ptr, int64_t weight_ptr, int64_t bias_ptr, int64_t out_ptr, int64_t workspace_size, 
                int64_t workspace_ptr, int64_t algo, std::vector<int> kernels, std::vector<int> paddings, std::vector<int> strides, 
                std::vector<int> dilations, int group,
                std::vector<int> in_shape, std::vector<int> weight_shape, std::vector<int> bias_shape, std::vector<int> out_shape, 
                std::string dtype, std::string layout, int64_t pstream) {

    if (!cudnn_handle) hipdnnCreate(&cudnn_handle);

    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t output_desc;
    hipdnnFilterDescriptor_t kernel_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
	setup_descriptor_nchw(kernels, paddings, strides, dilations, group, in_shape, weight_shape, bias_shape, out_shape, 
                dtype, layout,input_desc,output_desc, kernel_desc, conv_desc);


    size_t space_size_ = (size_t)workspace_size;
    hipdnnConvolutionFwdAlgo_t algo_ = (hipdnnConvolutionFwdAlgo_t)algo;

    float alpha_ = 1.0f;
    float beta_ = 0.0f;
    hipdnnStatus_t Error = hipdnnConvolutionForward(cudnn_handle, 
		&alpha_, input_desc,
		(void*)in_ptr, kernel_desc,
        (void*)weight_ptr, conv_desc,
		algo_, (void*)workspace_ptr,
		space_size_, &beta_,
		output_desc, (void*)out_ptr);
    // todo add bias


	if (Error != HIPDNN_STATUS_SUCCESS) {
		fprintf(stderr, "[Error] cudnn forward failed!\n");
	}

    return true;
}